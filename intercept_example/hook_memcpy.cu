#include <dlfcn.h>
#include <string.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <ctime>

#define STRINGIFY(x) STRINGIFY_AUX(x)
#define STRINGIFY_AUX(x) #x

/*
   1) use stringify wrap function name, so cuMemAlloc and cuMemAlloc_v2 can be all intercepted
   2) make sure function input types write, refer to cuda driver api https://docs.nvidia.com/cuda/cuda-driver-api/group__CUDA__MEM.html
   3) if function not intercepted succesfully, prob is caused that name of function is not called. e.g., cuMemcpy is not called, instead used cuMemcpyHtoD
*/
extern "C" {void *__libc_dlsym(void *map, const char *name);}
extern "C" {void *__libc_dlopen_mode(const char *name, int maddArgumentode);}

typedef void *(*fnDlsym)(void *, const char *);
static void *real_dlsym(void *handle, const char *symbol)
{
    static fnDlsym internal_dlsym = (fnDlsym)__libc_dlsym(__libc_dlopen_mode("libdl.so.2", RTLD_LAZY), "dlsym");
    return (*internal_dlsym)(handle, symbol);
}

typedef enum HookSymbolsEnum {
    SYM_CU_INIT,
    SYM_CU_MEM_ALLOC,
    SYM_CU_MEM_CPY_HTOD,
    SYM_CU_MEM_CPY,
    SYM_CU_SYMBOLS,
} HookSymbols;
static void* real_func[SYM_CU_SYMBOLS];

hipError_t hipInit(unsigned int flag) {
    std::cout << "====hipInit hooked====at ";
    std::chrono::time_point<std::chrono::system_clock> now = std::chrono::system_clock::now();
    auto duration = now.time_since_epoch();
    std::cout << duration.count() << std::endl;
    if (real_func[SYM_CU_INIT] == NULL) real_func[SYM_CU_INIT] = real_dlsym(RTLD_NEXT, "hipInit");
    return  ((hipError_t (*)(unsigned int))real_func[SYM_CU_INIT])(flag);
}

hipError_t hipMemcpyHtoD (hipDeviceptr_t dst, const void* srcHost, size_t ByteCount) {
    std::cout << "@@@@==hipMemcpyHtoD hooked=****===at ";
    std::chrono::time_point<std::chrono::system_clock> now = std::chrono::system_clock::now();
    auto duration = now.time_since_epoch();
    std::cout << duration.count() << std::endl;
    if (real_func[SYM_CU_MEM_CPY_HTOD] == NULL) real_func[SYM_CU_MEM_CPY_HTOD] = real_dlsym(RTLD_NEXT, "hipMemcpyHtoD");
    return  ((hipError_t (*)(hipDeviceptr_t,const void*,size_t))real_func[SYM_CU_MEM_CPY_HTOD])(dst, srcHost, ByteCount);
}
hipError_t cuMemcpy (hipDeviceptr_t dst, hipDeviceptr_t src, size_t ByteCount) {
    std::cout << "@@@@==hipMemcpy_ hooked=****===" << std::endl;
    if (real_func[SYM_CU_MEM_CPY] == NULL) real_func[SYM_CU_MEM_CPY] = real_dlsym(RTLD_NEXT, "hipMemcpy_");
    return  ((hipError_t (*)(hipDeviceptr_t,hipDeviceptr_t,size_t))real_func[SYM_CU_MEM_CPY])(dst, src, ByteCount);
}

hipError_t hipMalloc (hipDeviceptr_t* dptr, size_t bytesize)
{ 
    std::cout << "@@@@==hipMalloc hooked====" << std::endl;
    if (real_func[SYM_CU_MEM_ALLOC] == NULL) real_func[SYM_CU_MEM_ALLOC] = real_dlsym(RTLD_NEXT, "hipMalloc");
    return  ((hipError_t (*)(hipDeviceptr_t*, size_t))real_func[SYM_CU_MEM_ALLOC])(dptr, bytesize);
}

void *dlsym(void *handle, const char *symbol)   
{
    if (strcmp(symbol, STRINGIFY(hipMemcpyHtoD)) == 0) {
        if(real_func[SYM_CU_MEM_CPY_HTOD] == NULL) real_func[SYM_CU_MEM_CPY_HTOD] = real_dlsym(handle, symbol); 
        return (void*)(&hipMemcpyHtoD);
    }
    
    if (strcmp(symbol, STRINGIFY(hipInit)) == 0) {
        if(real_func[SYM_CU_INIT] == NULL) real_func[SYM_CU_INIT] = real_dlsym(handle, symbol);
        return (void*)(&hipInit);
    }
    
    if (strcmp(symbol, STRINGIFY(hipMalloc)) == 0) {
	if(real_func[SYM_CU_MEM_ALLOC] == NULL) real_func[SYM_CU_MEM_ALLOC] = real_dlsym(handle, symbol);
        return (void*)(&hipMalloc);
    }
    
    return (real_dlsym(handle, symbol));
}
