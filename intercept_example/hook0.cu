#include <dlfcn.h>
#include <string.h>
#include <iostream>
#include <hip/hip_runtime.h>

extern "C" {void *__libc_dlsym(void *map, const char *name);}
extern "C" {void *__libc_dlopen_mode(const char *name, int maddArgumentode);}

typedef void *(*fnDlsym)(void *, const char *);
static void *real_dlsym(void *handle, const char *symbol)
{
    static fnDlsym internal_dlsym = (fnDlsym)__libc_dlsym(__libc_dlopen_mode("libdl.so.2", RTLD_LAZY), "dlsym");
    return (*internal_dlsym)(handle, symbol);
}

static void *realFunctions;

hipError_t CUDAAPI hipInit(unsigned int flag) {
    hipError_t ret;
    std::cout << "func overlap works" << std::endl;
    if (realFunctions == NULL) realFunctions = real_dlsym(RTLD_NEXT, "hipInit");
    return  ((hipError_t (*)(unsigned int))realFunctions)(flag);
}

void *dlsym(void *handle, const char *symbol)   
{
    if (strcmp(symbol, "hipInit") == 0) {
        if(realFunctions == NULL) realFunctions = real_dlsym(handle, symbol); 
        return (void*)(&hipInit);
    }
    return (real_dlsym(handle, symbol));
}
